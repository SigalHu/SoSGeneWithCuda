#include <ctime>
#include <string>
#include "CudaRandUtils.cuh"

namespace CudaRandUtils{
	std::string getStatusStr(hiprandStatus_t status){
		switch (status){
		case HIPRAND_STATUS_SUCCESS:
			return "No errors.";
		case HIPRAND_STATUS_VERSION_MISMATCH:
			return "Header file and linked library version do not match.";
		case HIPRAND_STATUS_NOT_INITIALIZED:
			return "Generator not initialized.";
		case HIPRAND_STATUS_ALLOCATION_FAILED:
			return "Memory allocation failed.";
		case HIPRAND_STATUS_TYPE_ERROR:
			return "Generator is wrong type.";
		case HIPRAND_STATUS_OUT_OF_RANGE:
			return "Argument out of range.";
		case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
			return "Length requested is not a multple of dimension.";
		case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
			return "GPU does not have double precision required by MRG32k3a.";
		case HIPRAND_STATUS_LAUNCH_FAILURE:
			return "Kernel launch failure.";
		case HIPRAND_STATUS_PREEXISTING_FAILURE:
			return "Preexisting failure on library entry.";
		case HIPRAND_STATUS_INITIALIZATION_FAILED:
			return "Initialization of CUDA failed.";
		case HIPRAND_STATUS_ARCH_MISMATCH:
			return "Architecture mismatch, GPU does not support requested feature.";
		case HIPRAND_STATUS_INTERNAL_ERROR:
			return "Internal library error.";
		default:
			return "unrecognized error code.";
		}
	}

	bool createGenerator(hiprandGenerator_t &generator, hiprandRngType_t rng_type){
		hiprandStatus_t status = hiprandCreateGenerator(&generator, rng_type);
		if (HIPRAND_STATUS_SUCCESS != status){
			std::cerr << "#######################################" << std::endl;
			std::cerr << "hiprandCreateGenerator launch failed!" << std::endl;
			std::cerr << getStatusStr(status) << std::endl;
			std::cerr << "#######################################" << std::endl;
			return false;
		}

		status = hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
		if (HIPRAND_STATUS_SUCCESS != status){
			std::cerr << "#######################################" << std::endl;
			std::cerr << "hiprandSetPseudoRandomGeneratorSeed launch failed!" << std::endl;
			std::cerr << getStatusStr(status) << std::endl;
			std::cerr << "#######################################" << std::endl;
			return false;
		}
		return true;
	}

	bool generateNormal(thrust::device_vector<float> &dv, float mean, float stddev){
		hiprandGenerator_t generator;
		if (!createGenerator(generator, HIPRAND_RNG_PSEUDO_DEFAULT))
			return false;

		float *p = thrust::raw_pointer_cast(dv.data());
		unsigned int len = dv.size();
		hiprandStatus_t status = hiprandGenerateNormal(generator, p, len, mean, stddev);
		if (HIPRAND_STATUS_SUCCESS != status){
			std::cerr << "#######################################" << std::endl;
			std::cerr << "hiprandGenerateNormal launch failed!" << std::endl;
			std::cerr << getStatusStr(status) << std::endl;
			std::cerr << "#######################################" << std::endl;
			return false;
		}
		return true;
	}

	bool generateUniform(thrust::device_vector<float> &dv){
		hiprandGenerator_t generator;
		if (!createGenerator(generator, HIPRAND_RNG_PSEUDO_DEFAULT))
			return false;

		float *p = thrust::raw_pointer_cast(dv.data());
		unsigned int len = dv.size();
		hiprandStatus_t status = hiprandGenerateUniform(generator, p, len);
		if (HIPRAND_STATUS_SUCCESS != status){
			std::cerr << "#######################################" << std::endl;
			std::cerr << "hiprandGenerateUniform launch failed!" << std::endl;
			std::cerr << getStatusStr(status) << std::endl;
			std::cerr << "#######################################" << std::endl;
			return false;
		}
		return true;
	}
}
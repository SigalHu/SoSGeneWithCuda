#include <iostream>
#include "CudaCommonUtils.cuh"

namespace CudaCommonUtils{
	std::map<unsigned int, const dim3> getGridDimList();

	static std::map<unsigned int, const dim3> gridDimMap = CudaCommonUtils::getGridDimList();

	std::map<unsigned int, const dim3> getGridDimList(){
		int count = 0;
		hipDeviceProp_t prop;
		std::map<unsigned int, const dim3> map;

		hipError_t error = hipGetDeviceCount(&count);
		if (error != hipSuccess){
			std::cerr << "hipGetDeviceCount����ʧ�ܣ�" << std::endl;
			std::cerr << hipGetErrorString(error) << std::endl;
			return map;
		}

		for (unsigned int ii = 0; ii < count; ++ii){
			error = hipGetDeviceProperties(&prop, ii);
			if (error != hipSuccess){
				std::cerr << "hipGetDeviceProperties(ii = "<< ii << ")����ʧ�ܣ�" << std::endl;
				std::cerr << hipGetErrorString(error) << std::endl;
			}
			else
				map.emplace(ii, dim3(prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]));
		}
		return map;
	}

	const dim3 getGridDim(const unsigned int &dev_id){
		std::map<unsigned int, const dim3>::iterator iter = gridDimMap.find(dev_id);
		if (iter == gridDimMap.end()){
			dim3 defaultDim(65535, 65535, 65535);
			std::cerr << "dev_id = " << dev_id << "GPU�豸�����ڣ�" << std::endl;
			std::cerr << "ʹ��Ĭ��ֵ��dim3={" << defaultDim.x << ", " << defaultDim.y << ", " << defaultDim.z << "}" << std::endl;
			return defaultDim;
		}
		return iter->second;
	}

	void printDeviceProperties(){
		hipDeviceProp_t prop;
		hipError_t error = hipGetDeviceProperties(&prop, 0);
		if (hipSuccess != error){
			std::cerr << "hipGetDeviceProperties����ʧ�ܣ�" << std::endl;
			std::cerr << hipGetErrorString(error) << std::endl;
		}

		printf("###############################################\n");
		printf("Device Name : %s.\n", prop.name);
		printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
		printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
		printf("regsPerBlock : %d.\n", prop.regsPerBlock);
		printf("warpSize : %d.\n", prop.warpSize);
		printf("memPitch : %d.\n", prop.memPitch);
		printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
		printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("totalConstMem : %d.\n", prop.totalConstMem);
		printf("major.minor : %d.%d.\n", prop.major, prop.minor);
		printf("clockRate : %d.\n", prop.clockRate);
		printf("textureAlignment : %d.\n", prop.textureAlignment);
		printf("deviceOverlap : %d.\n", prop.deviceOverlap);
		printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
		printf("###############################################\n");
	}
}
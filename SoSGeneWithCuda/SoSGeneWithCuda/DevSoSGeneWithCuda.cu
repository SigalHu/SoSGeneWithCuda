#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include "DevSoSGeneWithCuda.cuh"

__global__ void cudaGaussianGene(float *dev_vec, unsigned int length, float *dev_uniform, unsigned int path_num,
	float omega_amp, float delta_alpha, float delta_omega, float delta_t, float sum_amp){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	float omega_n, phi_n, cos_sum;

	for (; y < length; y += y_step){
		cos_sum = 0;
		for (unsigned int xx = x; xx<path_num; xx += x_step){
			omega_n = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
			phi_n = 2 * CR_CUDART_PI*dev_uniform[xx];

			cos_value[threadIdx.x] = __cosf(omega_n * delta_t*y + phi_n);
			__syncthreads();

			for (unsigned int nn = (path_num <= x_step ? path_num : x_step) >> 1; nn > 0; nn >>= 1){
				if (x < nn){
					cos_value[threadIdx.x] += cos_value[threadIdx.x + nn];
				}
				__syncthreads();
			}

			if (x == 0 && y < length){
				cos_sum += cos_value[threadIdx.x];
			}
		}
		if (x == 0 && y < length){
			dev_vec[y] = sum_amp * cos_sum;
		}
	}
}

__global__ void cudaGaussianGeneIQ(float *dev_vec_i, float *dev_vec_q, unsigned int length, float *dev_uniform,
	unsigned int path_num, float omega_amp, float delta_alpha, float delta_omega, float delta_t, float sum_amp){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	__shared__ float sin_value[THREAD_NUM_PER_BLOCK];
	float omega_n_i, omega_n_q, phi_n_i, phi_n_q, cos_sum, sin_sum;

	for (; y < length; y += y_step){
		cos_sum = 0;
		sin_sum = 0;
		for (unsigned int xx = x; xx<path_num; xx += x_step){
			omega_n_i = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
			omega_n_q = omega_amp * __sinf(delta_alpha * xx) + delta_omega;
			phi_n_i = 2 * CR_CUDART_PI*dev_uniform[xx];
			phi_n_q = 2 * CR_CUDART_PI*dev_uniform[xx + path_num];

			cos_value[threadIdx.x] = __cosf(omega_n_i * delta_t*y + phi_n_i);
			sin_value[threadIdx.x] = __sinf(omega_n_q * delta_t*y + phi_n_q);
			__syncthreads();

			for (unsigned int nn = (path_num <= x_step ? path_num : x_step) >> 1; nn > 0; nn >>= 1){
				if (x < nn){
					cos_value[threadIdx.x] += cos_value[threadIdx.x + nn];
					sin_value[threadIdx.x] += sin_value[threadIdx.x + nn];
				}
				__syncthreads();
			}

			if (x == 0 && y < length){
				cos_sum += cos_value[threadIdx.x];
				sin_sum += sin_value[threadIdx.x];
			}
		}
		if (x == 0 && y < length){
			dev_vec_i[y] = sum_amp * cos_sum;
			dev_vec_q[y] = sum_amp * sin_sum;
		}
	}
}

__global__ void cudaLognormalGene(float *dev_vec, unsigned int length, float *dev_uniform, unsigned int path_num,
	float omega_amp, float delta_alpha, float delta_omega, float delta_t, float sum_amp){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	float omega_n, phi_n, cos_sum;

	for (; y < length; y += y_step){
		cos_sum = 0;
		for (unsigned int xx = x; xx<path_num; xx += x_step){
			omega_n = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
			phi_n = 2 * CR_CUDART_PI*dev_uniform[xx];

			cos_value[threadIdx.x] = __cosf(omega_n * delta_t*y + phi_n);
			__syncthreads();

			for (unsigned int nn = (path_num <= x_step ? path_num : x_step) >> 1; nn > 0; nn >>= 1){
				if (x < nn){
					cos_value[threadIdx.x] += cos_value[threadIdx.x + nn];
				}
				__syncthreads();
			}

			if (x == 0 && y < length){
				cos_sum += cos_value[threadIdx.x];
			}
		}
		if (x == 0 && y < length){
			dev_vec[y] = __expf(sum_amp * cos_sum);
		}
	}
}

__global__ void cudaNakagamiGene(float *dev_vec, unsigned int length, float *dev_uniform, 
	unsigned int path_num, float omega_amp, float delta_alpha, float delta_omega, float delta_t, float sum_amp, bool is_end){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	float omega_n, phi_n, cos_sum;

	for (; y < length; y += y_step){
		cos_sum = 0;
		for (unsigned int xx = x; xx<path_num; xx += x_step){
			omega_n = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
			phi_n = 2 * CR_CUDART_PI*dev_uniform[xx];

			cos_value[threadIdx.x] = __cosf(omega_n * delta_t*y + phi_n);
			__syncthreads();

			for (unsigned int mm = (path_num <= x_step ? path_num : x_step) >> 1; mm > 0; mm >>= 1){
				if (x < mm){
					cos_value[threadIdx.x] += cos_value[threadIdx.x + mm];
				}
				__syncthreads();
			}

			if (x == 0 && y < length){
				cos_sum += cos_value[threadIdx.x];
			}
		}
		if (x == 0 && y < length){
			if (is_end){
				dev_vec[y] = sqrtf(sum_amp*(dev_vec[y] + cos_sum*cos_sum));
			}
			else {
				dev_vec[y] += cos_sum * cos_sum;
			}
		}
		__threadfence_system();
	}
}

__global__ void cudaNakagamiGene2(float *dev_vec, unsigned int length, float *dev_uniform, unsigned int path_num,
	float omega_amp, float delta_alpha, float delta_omega, float delta_t, float sum_amp, unsigned int gaussian_n){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	float omega_n, phi_n, cos_sum, result;

	for (; y < length; y += y_step){
		result = 0;
		for (unsigned int nn = 0; nn < gaussian_n; nn++){
			cos_sum = 0;
			for (unsigned int xx = x; xx<path_num; xx += x_step){
				omega_n = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
				phi_n = 2 * CR_CUDART_PI*dev_uniform[xx + nn*path_num];

				cos_value[threadIdx.x] = __cosf(omega_n * delta_t*y + phi_n);
				__syncthreads();

				for (unsigned int mm = (path_num <= x_step ? path_num : x_step) >> 1; mm > 0; mm >>= 1){
					if (x < mm){
						cos_value[threadIdx.x] += cos_value[threadIdx.x + mm];
					}
					__syncthreads();
				}
				if (x == 0 && y < length){
					cos_sum += cos_value[threadIdx.x];
				}
			}
			if (x == 0 && y < length){
				result += cos_sum * cos_sum;
			}
		}
		if (x == 0 && y < length){
			dev_vec[y] = sqrtf(sum_amp * result);
		}
	}
}

__global__ void cudaNakagamiGeneIQ(float *dev_vec_i, float *dev_vec_q, unsigned int length, float *dev_uniform, unsigned int path_num,
	float omega_amp, float delta_alpha, float delta_omega, float delta_t, float sum_amp, unsigned int nak_m){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	__shared__ float sin_value[THREAD_NUM_PER_BLOCK];
	float omega_n_i, omega_n_q, phi_n_i, phi_n_q, cos_sum, sin_sum, result, phi;

	for (; y < length; y += y_step){
		result = 0;
		for (unsigned int nn = 0; nn < nak_m; nn++){
			cos_sum = 0;
			sin_sum = 0;
			for (unsigned int xx = x; xx<path_num; xx += x_step){
				omega_n_i = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
				omega_n_q = omega_amp * __sinf(delta_alpha * xx) + delta_omega;
				phi_n_i = 2 * CR_CUDART_PI*dev_uniform[xx + 2 * nn*path_num];
				phi_n_q = 2 * CR_CUDART_PI*dev_uniform[xx + (2 * nn + 1)*path_num];

				cos_value[threadIdx.x] = __cosf(omega_n_i * delta_t*y + phi_n_i);
				sin_value[threadIdx.x] = __sinf(omega_n_q * delta_t*y + phi_n_q);
				__syncthreads();

				for (unsigned int mm = (path_num <= x_step ? path_num : x_step) >> 1; mm > 0; mm >>= 1){
					if (x < mm){
						cos_value[threadIdx.x] += cos_value[threadIdx.x + mm];
						sin_value[threadIdx.x] += sin_value[threadIdx.x + mm];
					}
					__syncthreads();
				}
				if (x == 0 && y < length){
					cos_sum += cos_value[threadIdx.x];
					sin_sum += sin_value[threadIdx.x];
				}
			}
			if (x == 0 && y < length){
				result += cos_sum*cos_sum + sin_sum*sin_sum;
			}
		}
		if (x == 0 && y < length){
			result = sqrtf(sum_amp * result);
			phi = atanf(sin_sum / cos_sum) + (cos_sum < 0)*CR_CUDART_PI;
			dev_vec_i[y] = result *__cosf(phi);
			dev_vec_q[y] = result *__sinf(phi);
		}
	}
}

__global__ void cudaLogNakGene(float *dev_vec, unsigned int length, float *dev_uniform, unsigned int path_num, float mean,
	float omega_amp, float delta_alpha, float delta_omega, float delta_t, float log_sum_amp, float nak_sum_amp, unsigned int gaussian_n){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	float omega_n, phi_n, cos_sum, result;

	for (; y < length; y += y_step){
		result = 0;
		for (unsigned int nn = 0; nn < gaussian_n; nn++){
			cos_sum = 0;
			for (unsigned int xx = x; xx<path_num; xx += x_step){
				omega_n = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
				phi_n = 2 * CR_CUDART_PI*dev_uniform[xx + nn*path_num];

				cos_value[threadIdx.x] = __cosf(omega_n * delta_t*y + phi_n);
				__syncthreads();

				for (unsigned int mm = (path_num <= x_step ? path_num : x_step) >> 1; mm > 0; mm >>= 1){
					if (x < mm){
						cos_value[threadIdx.x] += cos_value[threadIdx.x + mm];
					}
					__syncthreads();
				}
				if (x == 0 && y < length){
					cos_sum += cos_value[threadIdx.x];
				}
			}
			if (x == 0 && y < length){
				if (nn < gaussian_n - 1){
					result += cos_sum * cos_sum;
				}
				else{
					dev_vec[y] = sqrtf(nak_sum_amp*result * __expf(log_sum_amp*cos_sum + mean));
				}
				
			}
		}
	}
}

__global__ void cudaLogNakGeneIQ(float *dev_vec_i, float *dev_vec_q, unsigned int length, float *dev_uniform, unsigned int path_num, float mean,
	float omega_amp, float delta_alpha, float delta_omega, float delta_t, float log_sum_amp, float nak_sum_amp, unsigned int nak_m){
	unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int x = tidx % path_num;
	unsigned int x_step = blockDim.x;

	unsigned int y = tidy;
	unsigned int y_step = gridDim.y;
	if (path_num < x_step){
		y = (y * x_step + tidx) / path_num;
		y_step *= x_step / path_num;
	}

	__shared__ float cos_value[THREAD_NUM_PER_BLOCK];
	__shared__ float sin_value[THREAD_NUM_PER_BLOCK];
	float omega_n_i, omega_n_q, phi_n_i, phi_n_q, cos_sum, sin_sum, result, phi;

	for (; y < length; y += y_step){
		result = 0;
		for (unsigned int nn = 0; nn < nak_m; nn++){
			cos_sum = 0;
			sin_sum = 0;
			for (unsigned int xx = x; xx<path_num; xx += x_step){
				omega_n_i = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
				omega_n_q = omega_amp * __sinf(delta_alpha * xx) + delta_omega;
				phi_n_i = 2 * CR_CUDART_PI*dev_uniform[xx + 2 * nn*path_num];
				phi_n_q = 2 * CR_CUDART_PI*dev_uniform[xx + (2 * nn + 1)*path_num];

				cos_value[threadIdx.x] = __cosf(omega_n_i * delta_t*y + phi_n_i);
				sin_value[threadIdx.x] = __sinf(omega_n_q * delta_t*y + phi_n_q);
				__syncthreads();

				for (unsigned int mm = (path_num <= x_step ? path_num : x_step) >> 1; mm > 0; mm >>= 1){
					if (x < mm){
						cos_value[threadIdx.x] += cos_value[threadIdx.x + mm];
						sin_value[threadIdx.x] += sin_value[threadIdx.x + mm];
					}
					__syncthreads();
				}
				if (x == 0 && y < length){
					cos_sum += cos_value[threadIdx.x];
					sin_sum += sin_value[threadIdx.x];
				}
			}
			if (x == 0 && y < length){
				result += cos_sum*cos_sum + sin_sum*sin_sum;
			}
		}

		phi = atanf(sin_sum / cos_sum) + (cos_sum < 0)*CR_CUDART_PI;

		cos_sum = 0;
		for (unsigned int xx = x; xx<path_num; xx += x_step){
			omega_n_i = omega_amp * __cosf(delta_alpha * xx) + delta_omega;
			phi_n_i = 2 * CR_CUDART_PI*dev_uniform[xx + 2 * nak_m*path_num];

			cos_value[threadIdx.x] = __cosf(omega_n_i * delta_t*y + phi_n_i);
			__syncthreads();

			for (unsigned int mm = (path_num <= x_step ? path_num : x_step) >> 1; mm > 0; mm >>= 1){
				if (x < mm){
					cos_value[threadIdx.x] += cos_value[threadIdx.x + mm];
				}
				__syncthreads();
			}
			if (x == 0 && y < length){
				cos_sum += cos_value[threadIdx.x];
			}
		}
		if (x == 0 && y < length){
			result = sqrtf(nak_sum_amp*result * __expf(log_sum_amp*cos_sum + mean));
			dev_vec_i[y] = result *__cosf(phi);
			dev_vec_q[y] = result *__sinf(phi);
		}
	}
}